#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <vector>
#include <chrono>

#include "../include/ecuda/event.hpp"
#include "../include/ecuda/array.hpp"
#include "../include/ecuda/matrix.hpp"

#include "config.hpp"
#ifndef BENCHMARK_THREADS
#define BENCHMARK_THREADS 480
#endif

typedef double value_type;

template<typename T,std::size_t N>
    __global__
void copyArray( typename ecuda::array<T,N>::const_kernel_argument src, typename ecuda::array<T,N>::kernel_argument dest )
{
    const int t = blockIdx.x*blockDim.x+threadIdx.x;
    if( t < src.size() ) dest[t] = src[t];
}

int main( int argc, char* argv[] )
{
    std::vector<int> dims = {50,100,200,500,1000,2000,5000,10000};
    for (int dim : dims) {
        ecuda::matrix<float> a(dim, 1), b(dim, 1);

        std::vector<float> a_cpu;
        a_cpu.resize(dim);
        static std::random_device rd;
        static std::mt19937 mt(rd());
        static std::uniform_real_distribution<> dist(-10, 10);
        for (int i =0;i<dim; ++i) {
            a_cpu[i] = dist(mt);
        }
        ecuda::copy(a_cpu.begin(), a_cpu.end(), a.begin());

        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
        for (int i=0; i<1000000; ++i) {
            ecuda::copy(a.begin(), a.end(), b.begin());
        }

        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        std::cout << "dim:" << dim << " time:" << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1000 << std::endl;
    }

    return EXIT_SUCCESS;

}

